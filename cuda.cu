#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

#include<bits/stdc++.h> 
#define MAX_ARGS 100000


extern "C" char ** words_array;

char ** parse_read(char * buffer, int * Length){
    char ** array;
    //array = (char **)calloc(MAX_ARGS, sizeof(char *));
    hipMallocManaged(&array, MAX_ARGS*sizeof(int));
    int len=0;
    
    // delimit commands by space and newline
    char * pch;
    
    
    pch = strtok (buffer,",\n \r\n");
    while (pch != NULL){
        *(array+len)=(char *)calloc(50, sizeof(char));
        strcpy(*(array+len), pch);
        //printf("psh: %s\n", pch);
        len++;
        pch = strtok (NULL, ",\n \r\n");
    }
    
    array[len]=NULL; //NULL terminate array for use in execv
    
#ifdef DEBUG_MODE
    for(int i=0; i<len;i++) {printf("debug: %s\n", (array)[i]);}
#endif
    *Length=len;
    return array;
}


static inline void cudaInitMaster(int rank, int nprocs, char * text, int * length){

    int cE, cudaDeviceCount;
	if((cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess ){
		printf(" Unable to determine cuda device count, error is %d, count is %d\n",
			cE, cudaDeviceCount );
		exit(-1);
	}
	if((cE = hipSetDevice( rank % cudaDeviceCount )) != hipSuccess ){
		printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
			rank, (rank % cudaDeviceCount), cE);
		exit(-1);
    }

    words_array = parse_read(text, length);

    
    //cudaMallocManaged(&dist_info.dp_array, num_files * num_files * tot_length*tot_length*sizeof(int));
	//cudaMallocManaged(&dist_info.dists, num_files*num_files*sizeof(int));
	

    
}


extern "C" void initMaster(int rank, int nprocs, char * text, int * length){
	cudaInitMaster(rank, nprocs, text, length);
}