#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<bits/stdc++.h> 

int count;

static inline void genInitMaster(){

    int cE, cudaDeviceCount;
	if((cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess ){
		printf(" Unable to determine cuda device count, error is %d, count is %d\n",
			cE, cudaDeviceCount );
		exit(-1);
	}
	if((cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess ){
		printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
			myrank, (myrank % cudaDeviceCount), cE);
		exit(-1);
    }
	
	count = 0;
   
}

__device__ checkSubstring(char* string, char* sub, int pos){
	
	for (int y = 0; y < strlen(sub); y++) {
		if (string[pos + y] == sub[y]) {
			continue;
		}
		else{
			return 0;
		}
	}
	return 1;

}

__global__ void countSubstring(char* string, char* sub) {

	for (int x = 0; x < (strlen(string) - strlen(sub)); x++) {
		count += checkSubstring(string, sub, x);
	}

}

extern "C" int getCount(){
	return count;
}