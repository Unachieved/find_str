#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

#include<bits/stdc++.h> 
#define MAX_ARGS 100000

extern "C" int count;
extern "C" char ** words_array;

static inline char ** parse_read(char * buffer, int * Length, int in_len){

    char ** array;
    //array = (char **)calloc(MAX_ARGS, sizeof(char *));
    hipMallocManaged(&array, MAX_ARGS*sizeof(int));
    int len=in_len;
    
    // delimit commands by space and newline
	char * pch;
    
    pch = strtok (buffer,",\n \r\n");
    while (pch != NULL){
        *(array+len)=(char *)calloc(50, sizeof(char));
        strcpy(*(array+len), pch);
        //printf("psh: %s\n", pch);
        len++;
        pch = strtok (NULL, ",\n \r\n");
    }
    
    array[len]=NULL; //NULL terminate array for use in execv
    *Length=len;
    return array;
}


static inline void cudaInitMaster(int rank, int nprocs, char * text, int * length, int in_len){

    int cE, cudaDeviceCount;
	if((cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess ){
		printf(" Unable to determine cuda device count, error is %d, count is %d\n",
			cE, cudaDeviceCount );
		exit(-1);
	}
	if((cE = hipSetDevice( rank % cudaDeviceCount )) != hipSuccess ){
		printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
			rank, (rank % cudaDeviceCount), cE);
		exit(-1);
    }
    words_array = parse_read(text, length, in_len);
   
}

extern "C" void initMaster(int rank, int nprocs, char * text, int * length, int in_len){
    cudaInitMaster(rank, nprocs, text, length, in_len);
}


__global__ void countSubstring(char** string, int length, char** to_find, int find, int * counter) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for(unsigned int i = index; i<length; i+=stride){
        
        int found = 1;

        if(i+(2*stride)<length){
            
            for(int j =0; j<find;j++){
                if(string[i+(j*stride)] == to_find[j])
                    found = 0;
            }
        }

        if(found)*counter++;
    }
}

extern "C" void kernelCall(char ** array, int length, ushort threadsCount, int numBlocks, char ** to_find, int find, int * counter){
	countSubstring<<<numBlocks, threadsCount>>>(array, length, to_find, find, counter);
}

