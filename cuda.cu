#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<bits/stdc++.h> 

int count = 0;

static inline void genInitMaster(){

    int cE, cudaDeviceCount;
	if((cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess ){
		printf(" Unable to determine cuda device count, error is %d, count is %d\n",
			cE, cudaDeviceCount );
		exit(-1);
	}
	if((cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess ){
		printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
			myrank, (myrank % cudaDeviceCount), cE);
		exit(-1);
    }

    int i = 0;
    char *p = strtok (buf, " ");
    char ** array = calloc(100, sizeof(char *));

    while (p != NULL)
    {
        (*array) = calloc(strlen(p), sizeof(char));
        array[i++] = p;
        p = strtok (NULL, " ");
    }

    
    //hipMallocManaged(&dist_info.dp_array, num_files * num_files * tot_length*tot_length*sizeof(int));
	//hipMallocManaged(&dist_info.dists, num_files*num_files*sizeof(int));
	
   
}

__device__ checkSubstring(char* string, char* sub, int pos){
	
	for (int y = 0; y < strlen(sub); y++) {
		if (str[pos + y] == sub[y]) {
			continue;
		}
		else{
			return 0;
		}
	}
	return 1;

}

__global__ void countSubstring(char* string, char* sub) {

	count = 0;

	for (int x = 0; x < (strlen(string) - strlen(sub)); x++) {
		count += checkSubstring(text, substring, x);
	}

}

extern "C" getCount(){
	return count;
}