#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<bits/stdc++.h> 
#define MAX_ARGS 100000

__global__ int* count;

extern "C" char ** words_array;

static inline char ** parse_read(char * buffer, int * Length, int in_len){

    char ** array;
    //array = (char **)calloc(MAX_ARGS, sizeof(char *));
    hipMallocManaged(&array, MAX_ARGS*sizeof(int));
    int len=in_len;
    
    // delimit commands by space and newline
	char * pch;
    
    
    pch = strtok (buffer,",\n \r\n");
    while (pch != NULL){
        *(array+len)=(char *)calloc(50, sizeof(char));
        strcpy(*(array+len), pch);
        //printf("psh: %s\n", pch);
        len++;
        pch = strtok (NULL, ",\n \r\n");
    }
    
    array[len]=NULL; //NULL terminate array for use in execv
    
#ifdef DEBUG_MODE
    for(int i=0; i<len;i++) {printf("debug: %s\n", (array)[i]);}
#endif
    *Length=len;
    return array;
}


static inline void cudaInitMaster(int rank, int nprocs, char * text, int * length, int in_len){

    int cE, cudaDeviceCount;
	if((cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess ){
		printf(" Unable to determine cuda device count, error is %d, count is %d\n",
			cE, cudaDeviceCount );
		exit(-1);
	}
	if((cE = hipSetDevice( rank % cudaDeviceCount )) != hipSuccess ){
		printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
			rank, (rank % cudaDeviceCount), cE);
		exit(-1);
    }
	
	*count = 0;
   
    words_array = parse_read(text, length, in_len);
   
}

extern "C" void initMaster(int rank, int nprocs, char * text, int * length, int in_len){
    cudaInitMaster(rank, nprocs, text, length, in_len);
}


__global__ void countSubstring(char** string, char* sub, int length) {

	for (int x = 0; x < length; x++) {
		if (strcmp(string[x], sub) == 0) {
			*count += 1;
		}
	}

}

extern "C" int* getCount(){
	return count;
}

extern "C" void kernelCall(char ** array, int length, ushort threadsCount, int numBlocks, char ** to_find, int find){
	countSubstring<<<numBlocks, threadsCount>>>(array, length, to_find, find);
}

