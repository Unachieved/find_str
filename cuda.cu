#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<bits/stdc++.h> 
#define MAX_ARGS 100000

__global__ int* count;

extern "C" char ** words_array;

char ** parse_read(char * buffer, int * Length){

    char ** array;
    //array = (char **)calloc(MAX_ARGS, sizeof(char *));
    hipMallocManaged(&array, MAX_ARGS*sizeof(int));
    int len=0;
    
    // delimit commands by space and newline
    char * pch;
    
    
    pch = strtok (buffer,",\n \r\n");
    while (pch != NULL){
        *(array+len)=(char *)calloc(50, sizeof(char));
        strcpy(*(array+len), pch);
        //printf("psh: %s\n", pch);
        len++;
        pch = strtok (NULL, ",\n \r\n");
    }
    
    array[len]=NULL; //NULL terminate array for use in execv
    
#ifdef DEBUG_MODE
    for(int i=0; i<len;i++) {printf("debug: %s\n", (array)[i]);}
#endif
    *Length=len;
    return array;
}


static inline void cudaInitMaster(int rank, int nprocs, char * text, int * length){

    int cE, cudaDeviceCount;
	if((cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess ){
		printf(" Unable to determine cuda device count, error is %d, count is %d\n",
			cE, cudaDeviceCount );
		exit(-1);
	}
	if((cE = hipSetDevice( rank % cudaDeviceCount )) != hipSuccess ){
		printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
			rank, (rank % cudaDeviceCount), cE);
		exit(-1);
    }
	
	*count = 0;
   
    words_array = parse_read(text, length);
   
}

extern "C" void initMaster(int rank, int nprocs, char * text, int * length){
    cudaInitMaster(rank, nprocs, text, length);
}


__global__ void countSubstring(char** string, char* sub, int length) {

	for (int x = 0; x < length; x++) {
		if (strcmp(string[x], sub) == 0) {
			*count += 1;
		}
	}

}

extern "C" int* getCount(){
	return count;
}

extern "C" void kernelCall(char ** array, int length, ushort threadsCount, int numBlocks){
	countSubstring<<<numBlocks, threadsCount>>>(array, length);
}

