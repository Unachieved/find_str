#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<bits/stdc++.h> 


static inline void genInitMaster(){

    int cE, cudaDeviceCount;
	if((cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess ){
		printf(" Unable to determine cuda device count, error is %d, count is %d\n",
			cE, cudaDeviceCount );
		exit(-1);
	}
	if((cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess ){
		printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
			myrank, (myrank % cudaDeviceCount), cE);
		exit(-1);
    }

    int i = 0;
    char *p = strtok (buf, " ");
    char ** array = calloc(100, sizeof(char *));

    while (p != NULL)
    {
        (*array) = calloc(strlen(p), sizeof(char));
        array[i++] = p;
        p = strtok (NULL, " ");
    }

    
    //hipMallocManaged(&dist_info.dp_array, num_files * num_files * tot_length*tot_length*sizeof(int));
	//hipMallocManaged(&dist_info.dists, num_files*num_files*sizeof(int));
	

    
}